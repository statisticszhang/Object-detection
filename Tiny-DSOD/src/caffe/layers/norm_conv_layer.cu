#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/norm_conv_layer.hpp"

namespace caffe {

template <typename Dtype>
void NormConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
   norm_layer_->Forward(norm_bottom_vec_, norm_top_vec_);
  const Dtype* weight = norm_weight_.gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
    }
  }
}

template <typename Dtype>
void NormConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = norm_weight_.gpu_data();
  Dtype* weight_diff = norm_weight_.mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
       // backward gradient from norm weight to weight
      norm_layer_->Backward(norm_top_vec_, propagate_down, norm_bottom_vec_);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(NormConvolutionLayer);

}  // namespace caffe
